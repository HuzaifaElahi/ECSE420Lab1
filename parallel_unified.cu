#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <chrono>
#include <>
#include <string.h>

#define AND 0
#define OR 1
#define NAND 2
#define NOR 3
#define XOR 4
#define NXOR 5
#define THREADS_PER_BLOCK 1024

inline hipError_t checkCudaErr(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA Runtime error at %s: %s\n", msg, hipGetErrorString(err));
    }
    return err;
}

__global__ void evaluateLogicGate(char* inputBuffer, int inputLength, char* outputBuffer, int inputLineLength, int outputLineLength) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;

    if (index < inputLength) {
        char* logicLine = inputBuffer + inputLineLength * (index);

        int operand1 = logicLine[0] - '0';
        int operand2 = logicLine[2] - '0';
        int gateType = logicLine[4] - '0';
        char result;

        switch (gateType) {
        case NOR:
            result = !(operand1 | operand2);
            break;
        case XOR:
            result = operand1 ^ operand2;
            break;
        case NAND:
            result = !(operand1 & operand2);
            break;
        case AND:
            result = operand1 & operand2;
            break;
        case OR:
            result = operand1 | operand2;
            break;
        case NXOR:
            result = !(operand1 ^ operand2);
            break;
        }

        char* outputLocation = outputBuffer + index * outputLineLength;
        char outputValue = result + '0';

        outputLocation[0] = outputValue;
        outputLocation[1] = '\0';
        outputLocation[2] = '\n';
    }
}

int main(int argc, char* argv[]) {

    if (argc != 4) {
        printf("Error: Please enter the input file path, input file length and output file path when running.\n");
        return 1;
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int inputLineLength = 7;
    int outputLineLength = 3;
    char* inputFileName = argv[1];
    int inputLength = atoi(argv[2]);
    char* outputFileName = argv[3];

    FILE* inputFile = fopen(inputFileName, "r");
    if (inputFile == NULL) {
        fprintf(stderr, "Error opening file.\n");
        return 1;
    }

    int inputSize = inputLength * inputLineLength * sizeof(unsigned char);
    int outputSize = inputLength * outputLineLength * sizeof(unsigned char);

    char* inputBuffer = (char*)malloc(inputSize);

    char buf[7];
    int addressLocation = 0;
    while (fgets(buf, sizeof buf, inputFile) != NULL) {
        strcpy(inputBuffer + addressLocation, buf);
        addressLocation += inputLineLength;
    }

    fclose(inputFile);

    char* cudaBuffer;
    char* outputBuffer;

    hipMallocManaged(&cudaBuffer, inputSize);
    hipMallocManaged(&outputBuffer, outputSize);

    for (int i = 0; i < inputSize; i++) {
        cudaBuffer[i] = inputBuffer[i];
    }

    hipEventRecord(start);
    evaluateLogicGate<<< (inputLength + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >> > (cudaBuffer, inputLength, outputBuffer, inputLineLength, outputLineLength);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    checkCudaErr(hipDeviceSynchronize(), "Syncronization");
    checkCudaErr(hipGetLastError(), "GPU");

    float duration = 0;
    hipEventElapsedTime(&duration, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    FILE* outputFile = fopen(outputFileName, "w");
    int counter = 0;
    addressLocation = 0;

    while (counter < inputLength) {
        if (counter == inputLength - 1) {
            fprintf(outputFile, "%s", outputBuffer + addressLocation);
        }
        else {
            fprintf(outputFile, "%s\n", outputBuffer + addressLocation);
        }
        addressLocation += outputLineLength;
        counter++;
    }

    fclose(outputFile);

    printf("Completed!\n");
    printf("Time: %.6fms\n", duration);

    return 0;
}